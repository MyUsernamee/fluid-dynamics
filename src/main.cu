#include "hip/hip_runtime.h"
#include "raylib.h"
#include <vector>
#include <glm/glm.hpp>
#include <algorithm>
#include <string.h>
#include <functional>
#include "simulation.hpp"
#include "config.hpp"
#include <stdio.h>
#include <iostream>

#define AUDIO_BUFFER_SIZE 1024

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

enum DrawMode
{

    COLOR = 0,
    PRESSURE = 1,
    VELOCITY = 2,

};

void drawArrow(glm::vec2 direction, int x, int y, int w)
{
    DrawLineEx({(float)x, (float)y}, {x + direction.x, y + direction.y}, w, WHITE);
}

template <typename T>
std::pair<T, T> min_max(std::function<T(int index)> f, int count)
{

    T min = f(0);
    T max = f(0);

    for (int i = 0; i < count; ++i)
    {
        T value = f(i);

        min = std::min(min, value);
        max = std::max(max, value);
    }

    return {min, max};
}

void draw(Grid<FluidData, N, N> data, DrawMode mode)
{
    std::pair<float, float> lim;
    float min;
    float max;
    lim = min_max<float>([&](int i)
                         { return data.data[i].getPressure(dx * dx); }, N * N);

    max = lim.second;
    min = lim.first;

    if (mode == PRESSURE)
    {
        // Draw a little box to show limits
        DrawRectangleLines(10, 9, 40, 81, WHITE);
        for (int i = 0; i < 80; ++i)
        {
            DrawLine(11, i + 10, 49, i + 10, ColorFromHSV(250.0 - std::fmod((80.0 - i) / 80.0 * 250.0, 360.0), 1.0, 1.0));
        }

        DrawText(TextFormat("%f", max), 60, 9, 12, WHITE);
        DrawText(TextFormat("%f", min), 60, 81, 12, WHITE);
    }

    for (int x = 0; x < N; ++x)
    {

        // DrawLine(start_x + x * cell_size, 0, start_x + x * cell_size, HEIGHT, WHITE);

        for (int y = 0; y < N; ++y)
        {

            Color color = BLACK;

            glm::vec3 c;
            glm::vec2 v;

            switch (mode)
            {

            case PRESSURE:

                color = ColorFromHSV(std::fmod(250.0 - (data.get(x, y).getPressure(dx * dx) - min) / (max - min) * 250.0, 360.0), 1.0, 1.0);

                // We are going to draw a handy diagram for helping visualize

                break;

            case COLOR:

                c = glm::clamp(data.get(x, y).color, glm::vec3(0.0), glm::vec3(1.0));

                color = ColorFromNormalized({c.x, c.y, c.z, 1.0});

                break;

            case VELOCITY:

                v = glm::clamp(glm::abs(data.get(x, y).vel / data.get(x, y).density / 4.0f), glm::vec2(0.0), glm::vec2(1.0));
                color = ColorFromNormalized({v.x, v.y, 0.0, 1.0});

                break;
            }

            // drawArrow(data.get(x, y).vel * cell_size, start_x + x * cell_size + 0.5 * cell_size, start_y + y * cell_size + 0.5 * cell_size, 1.0);
            DrawRectangle(start_x + x * cell_size + 0.5 * cell_size, start_y + y * cell_size + 0.5 * cell_size, cell_size, cell_size,
                          color);

            if (data.get(x, y).wall)
            {
                DrawRectangle(start_x + x * cell_size + 0.5 * cell_size, start_y + y * cell_size + 0.5 * cell_size, cell_size, cell_size, RED);
            }
        }
    }
}

__global__ void update(Grid<FluidData, N, N> grid, Grid<FluidData, N, N> new_grid, float dt)
{

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= N - 1 || y >= N - 1 || x < 0 || y < 0)
        return;

    if (grid.get(x, y).wall)
        return;

    applyPressureForce(grid, dt, x, y);
    advect(grid, new_grid, dt, x, y);
    // integrate(new_grid, dt, x, y);
    grid.set(x, y, new_grid.get(x, y));
}

int main()
{

    InitWindow(WIDTH, HEIGHT, "Fluid Dynamics");
    InitAudioDevice();

    dim3 blocks(N / 16 + 1, N / 16 + 1);
    dim3 threads(16, 16);

    Grid<FluidData, N, N> grid(FluidData(ATM_PRESSURE, ATM_TEMP));

    // Make edges walls
    for (int i = 0; i < N; ++i)
    {
        grid.get(i, 0).wall = true;
        grid.get(i, N - 1).wall = true;
        grid.get(0, i).wall = true;
        grid.get(N - 1, i).wall = true;

        grid.get(i, 0).density = 0.0;
        grid.get(i, N - 1).density = 0.0;
        grid.get(0, i).density = 0.0;
        grid.get(N - 1, i).density = 0.0;
    }

    Grid<FluidData, N, N> d_grid = Grid<FluidData, N, N>(FluidData());
    Grid<FluidData, N, N> d_back_grid = Grid<FluidData, N, N>(FluidData());
    Grid<float, N, N> d_p_grid(0.0);

    delete d_grid.data; // We don't need another array on the cpu
    delete d_back_grid.data;
    delete d_p_grid.data;

    hipMalloc(&d_grid.data, sizeof(FluidData) * N * N);
    hipMalloc(&d_back_grid.data, sizeof(FluidData) * N * N);
    hipMalloc(&d_p_grid.data, sizeof(float) * N * N);

    // SetTargetFPS(60);

    auto view_mode = PRESSURE;
    auto mouse_size = 10;
    short *data = new short[AUDIO_BUFFER_SIZE];
    int data_index = 0;
    float *d_o;
    hipMalloc(&d_o, sizeof(float) * AUDIO_BUFFER_SIZE + 1);
    float *o = new float[AUDIO_BUFFER_SIZE + 1];

    while (!WindowShouldClose())
    {
        BeginDrawing();

        ClearBackground(BLACK);

        DrawFPS(10, 200);

        auto mouse_p = getLocalPosition(GetMouseX(), GetMouseY());
        auto mouse_d = GetMouseDelta();
        draw(grid, view_mode);

        // Set the pressure of the cells near the edge but not the edge edge to atmospheric pressure
        for (int i = 0; i < N - 1; ++i)
        {
            grid.get(1, i + 1).setPressure(ATM_PRESSURE);
            grid.get(N - 2, i + 1).setPressure(ATM_PRESSURE);
            grid.get(i + 1, 1).setPressure(ATM_PRESSURE);
            grid.get(i + 1, N - 2).setPressure(ATM_PRESSURE);
        }

        if (IsMouseButtonDown(MOUSE_BUTTON_LEFT))
        {
            // Make Wall
            fillCircle(grid, FluidData{glm::vec2(0.0), glm::vec3(0.0), true, 0.0}, mouse_p.x, mouse_p.y, mouse_size);
        }
        if (IsMouseButtonDown(MOUSE_BUTTON_RIGHT))
        {
            fillCircle(grid, FluidData{(glm::vec2(mouse_d.x + 0.0001, mouse_d.y)) * 1.0f, glm::vec3(1.0, 0.0, 0.0), false, grid.get(mouse_p.x, mouse_p.y).getPressure(dx * dx) * 1.1f}, mouse_p.x, mouse_p.y, mouse_size);
        }
        if (IsMouseButtonDown(MOUSE_BUTTON_MIDDLE))
        {
            fillCircle(grid, FluidData{glm::vec2(0.0), glm::vec3(0.0), false, grid.get(mouse_p.x, mouse_p.y).getPressure(dx * dx) / 1.1f}, mouse_p.x, mouse_p.y, mouse_size);
        }

        // Just change color
        if (IsKeyPressed(KEY_R))
        {
            setCircleColor(grid, glm::vec3(1.0), mouse_p.x, mouse_p.y, mouse_size);
        }

        drawArrow(getDataAtPoint(grid, mouse_p.x, mouse_p.y).vel * cell_size, GetMouseX(), GetMouseY(), 4);
        auto a = getGlobalPosition(mouse_p.x, mouse_p.y);
        DrawCircle(a.x, a.y, 4.0, RED);

        // Render rect lines to show mouse size and scale
        DrawRectangleLines(GetMouseX() - mouse_size / 2 * cell_size, GetMouseY() - mouse_size / 2 * cell_size, mouse_size * cell_size, mouse_size * cell_size, WHITE);

        // Draw text
        DrawText(TextFormat("View Mode: %d", view_mode), 10, 100, 12, WHITE);
        DrawText(TextFormat("Mouse Size: %d", mouse_size), 10, 130, 12, WHITE);

        mouse_size += GetMouseWheelMove();

        if (IsKeyPressed(KEY_SPACE))
        {
            view_mode = (DrawMode)((view_mode + 1) % 3);
        }

        // Update grid

        gpuErrchk(hipMemcpy(d_grid.data, grid.data, sizeof(FluidData) * N * N, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_back_grid.data, d_grid.data, sizeof(FluidData) * N * N, hipMemcpyDeviceToDevice));

        for (int i = 0; i < 400; ++i)
        {
            update<<<blocks, threads>>>(d_grid, d_back_grid, 1.0 / 60.0 / 400.0);
        }
        gpuErrchk(hipMemcpy(grid.data, d_back_grid.data, sizeof(FluidData) * N * N, hipMemcpyDeviceToHost));

        EndDrawing();
    }

    CloseWindow();
}